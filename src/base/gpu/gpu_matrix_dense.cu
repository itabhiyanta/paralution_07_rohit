#include "hip/hip_runtime.h"
// *************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2012-2014 Dimitar Lukarski
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// *************************************************************************



// PARALUTION version 0.7.0 


#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_coo.hpp"
#include "gpu_matrix_dia.hpp"
#include "gpu_matrix_ell.hpp"
#include "gpu_matrix_hyb.hpp"
#include "gpu_matrix_mcsr.hpp"
#include "gpu_matrix_bcsr.hpp"
#include "gpu_matrix_dense.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_dense.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_dense.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <assert.h>
#include <hip/hip_runtime.h>


namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::GPUAcceleratorMatrixDENSE() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::GPUAcceleratorMatrixDENSE(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixDENSE::GPUAcceleratorMatrixDENSE()",
            "constructor with local_backend");

  this->mat_.val = NULL;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::~GPUAcceleratorMatrixDENSE() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixDENSE::~GPUAcceleratorMatrixDENSE()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixDENSE<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::AllocateDENSE(const int nrow, const int ncol) {

  assert( ncol  >= 0);
  assert( nrow  >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nrow*ncol > 0) {

    allocate_gpu(nrow*ncol, &this->mat_.val);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nrow*ncol, mat_.val);   

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nrow*ncol;

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.val);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixDENSE<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateDENSE(this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {
      
      hipMemcpy(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) { 

        hipMemcpy(this->mat_.val,         // dst
                   gpu_cast_mat->mat_.val, // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixDENSE<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateDENSE(dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

        hipMemcpy(gpu_cast_mat->mat_.val, // dst
                   this->mat_.val,         // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToHost);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFromHostAsync(const HostMatrix<ValueType> &src) {

  const HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpyAsync(this->mat_.val,     // dst
                      cast_mat->mat_.val, // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyToHostAsync(HostMatrix<ValueType> *dst) const {

  HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixDENSE<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateDENSE(this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {
      
      hipMemcpyAsync(cast_mat->mat_.val, // dst
                      this->mat_.val,     // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFromAsync(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) { 

      hipMemcpy(this->mat_.val,         // dst
                 gpu_cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHostAsync(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyToAsync(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixDENSE<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateDENSE(dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

        hipMemcpy(gpu_cast_mat->mat_.val, // dst
                   this->mat_.val,         // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToHost);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHostAsync(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool GPUAcceleratorMatrixDENSE<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixDENSE<ValueType>   *cast_mat_dense;
  
  if ((cast_mat_dense = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_dense);
      return true;

  }

  /*
  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->Clear();
    
    FATAL_ERROR(__FILE__, __LINE__);
    
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = cast_mat_csr->get_nnz();
    
    return true;
    
  }
  */

  return false;

}

template <>
void GPUAcceleratorMatrixDENSE<float>::Apply(const BaseVector<float> &in, BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    
    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in) ; 
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;
    hipblasOperation_t trans;
    trans=HIPBLAS_OP_N;

    const float alpha = 1.0f, beta = 0.0f;
    stat_t = hipblasSgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), trans, 
                         this->get_nrow(), this->get_ncol(),
                         &alpha,
                         this->mat_.val, this->get_nrow(), 
                         cast_in->vec_, 1, 
                         &beta, cast_out->vec_, 1);
    
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);


  }

}

template <>
void GPUAcceleratorMatrixDENSE<double>::Apply(const BaseVector<double> &in, BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    
    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in) ; 
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;
    hipblasOperation_t trans;
    trans=HIPBLAS_OP_N;

    const double alpha = 1.0f, beta = 0.0f;
    stat_t = hipblasDgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), trans, 
                         this->get_nrow(), this->get_ncol(),
                         &alpha,
                         this->mat_.val, this->get_nrow(), 
                         cast_in->vec_, 1, 
                         &beta, cast_out->vec_, 1);
    
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);


  }

}

template <>
void GPUAcceleratorMatrixDENSE<float>::ApplyAdd(const BaseVector<float> &in, const float scalar,
                                                BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    
    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in) ; 
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;
    hipblasOperation_t trans;
    trans=HIPBLAS_OP_N;

    const float alpha = scalar, beta = 0.0f;
    stat_t = hipblasSgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), trans, 
                         this->get_nrow(), this->get_ncol(),
                         &alpha,
                         this->mat_.val, this->get_nrow(), 
                         cast_in->vec_, 1, 
                         &beta, cast_out->vec_, 1);
    
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);


  }

}

template <>
void GPUAcceleratorMatrixDENSE<double>::ApplyAdd(const BaseVector<double> &in, const double scalar,
                                                 BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    
    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in) ; 
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;
    hipblasOperation_t trans;
    trans=HIPBLAS_OP_N;

    const double alpha = scalar, beta = 0.0f;
    stat_t = hipblasDgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), trans, 
                         this->get_nrow(), this->get_ncol(),
                         &alpha,
                         this->mat_.val, this->get_nrow(), 
                         cast_in->vec_, 1, 
                         &beta, cast_out->vec_, 1);
    
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);


  }

}



template class GPUAcceleratorMatrixDENSE<double>;
template class GPUAcceleratorMatrixDENSE<float>;

}

